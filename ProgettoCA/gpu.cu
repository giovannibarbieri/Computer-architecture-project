#include <iostream>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

void readList(long**);
void mergesort(long*);
__global__ void gpu_mergesort(long*, long*, long, long);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);

#define min(a, b) (a < b ? a : b)
#define size 1000000
#define NumThreads 1024
#define NumBlocks 65535

int main(int argc, char** argv) {

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    long *data;
    readList(&data);

    hipEventRecord(start);

    mergesort(data);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Tempo di esecuzione: %f secondi\n", float(elapsedTime/1000));

    hipEventDestroy(start);
    hipEventDestroy(stop);

}

void mergesort(long* data){
    long* D_data;
    long* D_swp;

    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));

    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = NumThreads * NumBlocks;


    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<NumBlocks, NumThreads>>>(A, B, width, slices);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
}

__global__ void gpu_mergesort(long* source, long* dest, long width, long slices) {
    unsigned int idx = threadIdx.x + blockIdx.x*NumThreads;
    long start = width*idx*slices,
         middle,
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;
             middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long __align__ (8) i = start;
    long __align__ (8) j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

typedef struct {
    int v;
    void* next;
} LinkNode;

void readList(long** list) {
    LinkNode* node = 0;
    LinkNode* first = 0;
    for (long v=size; v>0; v--) {
        LinkNode* next = new LinkNode();
        next->v = v;
        if (node)
            node->next = next;
        else
            first = next;
        node = next;
    }

    *list = new long[size];
    node = first;
    long i = 0;
    while (node) {
       (*list)[i++] = node->v;
       node = (LinkNode*) node->next;
    }
}